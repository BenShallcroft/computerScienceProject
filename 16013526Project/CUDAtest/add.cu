

//In theory, GPU accelerated code



#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

using namespace std;

__global__ //Kernel function to add the elements of two arrays
void add(int n, float *x, float *y)
{	
	for(int i= 0; i < n; i++) y[i] = x[i] + y[i]; //Note: i is now the thread index, and each loop through changes to next thread in the block
}

int main(void)
{
	int N = 1<<20;
	
	float *x, *y;
	
	//Allovate unified memory - accessible from CPU or GPU
	hipMallocManaged(&x, N*sizeof(float));
	hipMallocManaged(&y, N*sizeof(float));
	
	//Initiallise the x and y arrays on the host
	for(int i= 0; i < N; i++)
	{
		x[i] = 1.0f;
		y[i] = 2.0f;
	}
	
	//Run kernel on 1M elements on the GPU
	add<<<1, 1>>>(N, x, y);
	
	//Wait for GPU to finish before accessing on host
	hipDeviceSynchronize();
	
	float maxError = 0.0f;
	
	for(int i=0; i < N; i++) maxError = fmax(maxError, fabs(y[i] -3.0f));
	
	cout << "Max error: " << maxError << endl;
	
	//Free memory
	hipFree(x);
	hipFree(y);
	
	return 0;
}
