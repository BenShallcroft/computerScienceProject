

//In theory, GPU accelerated code



#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

using namespace std;

__global__ //Kernel function to add the elements of two arrays
void add(int n, float *x, float *y)
{
	//int index = threadIdx.x; //Contains index of current thread within it's block
	//int stride = blockDim.x; //Number of threads in the block
	
	//gridDim.x is number of blocks in the grid
	//blockId.x contains the index of the current thread block on the grid
	
	int index = blockIdx.x * blockDim.x + threadIdx.x;
		//Each thread get's it's index by computing the offset to the beginning of it's block and adding the threads index within the block.
	int stride = blockDim.x * gridDim.x;
		//Stride is the total number of threads in the grid.
	
	
	for(int i= index; i < n; i += stride) y[i] = x[i] + y[i]; //Note: i is now the thread index, and each loop through changes to next thread in the block
}

int main(void)
{
	int N = 1<<20;
	
	float *x, *y;
	
	//Allovate unified memory - accessible from CPU or GPU
	hipMallocManaged(&x, N*sizeof(float));
	hipMallocManaged(&y, N*sizeof(float));
	
	//Initiallise the x and y arrays on the host
	for(int i= 0; i < N; i++)
	{
		x[i] = 1.0f;
		y[i] = 2.0f;
	}
	
	//Calculate the number of blocks of parallel threads to launch
	int blockSize = 256;
	int numBlocks = (N + blockSize - 1) / blockSize;
	
	//Run kernel on 1M elements on the GPU
		//CUDA GPU's run kernels using blocks of threads that are a multiple of 32 in size, so 256 threads is a reasonable size to choose
	add<<<numBlocks, blockSize>>>(N, x, y);
	
	//Wait for GPU to finish before accessing on host
	hipDeviceSynchronize();
	
	float maxError = 0.0f;
	
	for(int i=0; i < N; i++) maxError = fmax(maxError, fabs(y[i] -3.0f));
	
	cout << "Max error: " << maxError << endl;
	
	//Free memory
	hipFree(x);
	hipFree(y);
	
	return 0;
}
